#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2013, NVIDIA CORPORATION.  All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" 
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE 
 * ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 *
 * Code and text by Sean Baxter, NVIDIA Research
 * See http://nvlabs.github.io/moderngpu for repository and documentation.
 *
 ******************************************************************************/

#include "kernels/mergesort.cuh"

using namespace mgpu;

// Copy MergesortKeys host function from kernels/mergesort.cuh and modify to
// expose the tuning structure.
template<typename Tuning, typename T, typename Comp>
MGPU_HOST void TuningMergesortKeys(T* data_global, int count, Comp comp,
	CudaContext& context) {
	
	int2 launch = Tuning::GetLaunchParams(context);
	
	const int NV = launch.x * launch.y;
	int numBlocks = MGPU_DIV_UP(count, NV);
	int numPasses = FindLog2(numBlocks, true);

	MGPU_MEM(T) destDevice = context.Malloc<T>(count);
	T* source = data_global;
	T* dest = destDevice->get();

	KernelBlocksort<Tuning, false>
		<<<numBlocks, launch.x, 0, context.Stream()>>>(source, (const int*)0,
		count, (1 & numPasses) ? dest : source, (int*)0, comp);
	if(1 & numPasses) std::swap(source, dest);

	for(int pass = 0; pass < numPasses; ++pass) {
		int coop = 2<< pass;
		MGPU_MEM(int) partitionsDevice = MergePathPartitions<MgpuBoundsLower>(
			source, count, source, 0, NV, coop, comp, context);
		
		KernelMerge<Tuning, false, true>
			<<<numBlocks, launch.x, 0, context.Stream()>>>(source, 
			(const int*)0, count, source, (const int*)0, 0, 
			partitionsDevice->get(), coop, dest, (int*)0, comp);
		std::swap(dest, source);
	}
}

template<typename Tuning, typename T>
void BenchmarkTunedMergesort(int count, int numIt, CudaContext& context) {
	MGPU_MEM(T) source = context.GenRandom<T>(count, 0, (T)count);
	MGPU_MEM(T) data = context.Malloc<T>(count);
	std::vector<T> sourceHost;
	source->ToHost(sourceHost);

	double mgpuElapsed = 0;
	for(int it = 0; it < numIt; ++it) {
		source->ToDevice(data->get(), count);
		context.Start();
		TuningMergesortKeys<Tuning>(data->get(), count, mgpu::less<T>(),
			context);
		mgpuElapsed += context.Split();
	}
	
	double bytes = 2 * sizeof(T) * count;
	double mgpuThroughput = (double)count * numIt / mgpuElapsed;
	double mgpuBandwidth = bytes * numIt / mgpuElapsed;

	int2 launch = Tuning::GetLaunchParams(context);
	printf("%3dx%2d - %s: %9.3lf M/s  %7.3lf GB/s\n",
		launch.x, launch.y, FormatInteger(count).c_str(),
		mgpuThroughput / 1.0e6, mgpuBandwidth / 1.0e9);
	
	// Verify
	std::sort(sourceHost.begin(), sourceHost.end());
	std::vector<T> host;
	data->ToHost(host);
	for(int i = 0; i < count; ++i)
		if(sourceHost[i] != host[i]) {
			printf("MISMATCH AT %d\n", i);
			exit(0);
		}
}

int main(int argc, char** argv) {
	ContextPtr context = CreateCudaDevice(argc, argv, true);

	const int N = 10000000;

	typedef LaunchBoxVT<128, 5, 0> Tuning1;
	typedef LaunchBoxVT<128, 7, 0> Tuning2;
	typedef LaunchBoxVT<128, 11, 0> Tuning3;
	typedef LaunchBoxVT<128, 15, 0> Tuning4;
	typedef LaunchBoxVT<128, 19, 0> Tuning5;
	typedef LaunchBoxVT<128, 23, 0> Tuning6;
	typedef LaunchBoxVT<128, 27, 0> Tuning7;
	typedef LaunchBoxVT<256, 5, 0> Tuning8;
	typedef LaunchBoxVT<256, 7, 0> Tuning9;
	typedef LaunchBoxVT<256, 11, 0> Tuning10;
	typedef LaunchBoxVT<256, 15, 0> Tuning11;
	typedef LaunchBoxVT<256, 19, 0> Tuning12;
	typedef LaunchBoxVT<256, 23, 0> Tuning13;
	typedef LaunchBoxVT<256, 27, 0> Tuning14;

	typedef int T1;
	typedef int64 T2;

	printf("Tuning for type %s.\n", TypeIdName<T1>());
	BenchmarkTunedMergesort<Tuning1, T1>(N, 200, *context);
	BenchmarkTunedMergesort<Tuning2, T1>(N, 200, *context);
	BenchmarkTunedMergesort<Tuning3, T1>(N, 200, *context);
	BenchmarkTunedMergesort<Tuning4, T1>(N, 200, *context);
	BenchmarkTunedMergesort<Tuning5, T1>(N, 200, *context);
	BenchmarkTunedMergesort<Tuning6, T1>(N, 200, *context);
	BenchmarkTunedMergesort<Tuning7, T1>(N, 200, *context);
	BenchmarkTunedMergesort<Tuning8, T1>(N, 200, *context);
	BenchmarkTunedMergesort<Tuning9, T1>(N, 200, *context);
	BenchmarkTunedMergesort<Tuning10, T1>(N, 200, *context);
	BenchmarkTunedMergesort<Tuning11, T1>(N, 200, *context);
	BenchmarkTunedMergesort<Tuning12, T1>(N, 200, *context);
	
	printf("\nTuning for type %s.\n", TypeIdName<T2>());
	BenchmarkTunedMergesort<Tuning1, T2>(N, 200, *context);
	BenchmarkTunedMergesort<Tuning2, T2>(N, 200, *context);
	BenchmarkTunedMergesort<Tuning3, T2>(N, 200, *context);
	BenchmarkTunedMergesort<Tuning4, T2>(N, 200, *context);
	BenchmarkTunedMergesort<Tuning5, T2>(N, 200, *context);
	BenchmarkTunedMergesort<Tuning6, T2>(N, 200, *context);
	BenchmarkTunedMergesort<Tuning7, T2>(N, 200, *context);
	BenchmarkTunedMergesort<Tuning8, T2>(N, 200, *context);
	BenchmarkTunedMergesort<Tuning9, T2>(N, 200, *context);
	BenchmarkTunedMergesort<Tuning10, T2>(N, 200, *context);
	BenchmarkTunedMergesort<Tuning11, T2>(N, 200, *context);
	BenchmarkTunedMergesort<Tuning12, T2>(N, 200, *context);
}
