#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2013, NVIDIA CORPORATION.  All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" 
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE 
 * ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 *
 * Code and text by Sean Baxter, NVIDIA Research
 * See http://nvlabs.github.io/moderngpu for repository and documentation.
 *
 ******************************************************************************/

#include "kernels/segreducecsr.cuh"

using namespace mgpu;

enum TestType {
	TestTypeNormal,
	TestTypeIndirect,
	TestTypePreprocess
};


template<typename T>
void TestCsrReduce(int count, int randomSize, int numIterations,
	TestType testType, bool supportEmpty, CudaContext& context) {
		 
#ifdef _DEBUG
	numIterations = 1;
#endif
	
	std::vector<int> segCountsHost, csrHost;
	int total = 0;
	int numValidRows = 0;
	while(total < count) {
		int randMin = supportEmpty ? 0 : 1;
		int segSize = Rand(randMin, min(randomSize, count - total));
		numValidRows += 0 != segSize;
		csrHost.push_back(total ? (csrHost.back() + segCountsHost.back()) : 0);
		segCountsHost.push_back(segSize);
		total += segSize;
	}
	int numRows = (int)segCountsHost.size();
	
	std::vector<int> sourcesHost(numRows);
	for(int i = 0; i < numRows; ++i)
		sourcesHost[i] = Rand(0, max(0, count - randomSize));

	MGPU_MEM(int) csrDevice = context.Malloc(csrHost);
	MGPU_MEM(int) sourcesDevice = context.Malloc(sourcesHost);

	// Generate random ints as input.
	std::vector<T> dataHost(count);
	for(int i = 0; i < count; ++i)
		dataHost[i] = (T)Rand(1, 9);
	MGPU_MEM(T) dataDevice = context.Malloc(dataHost);

	MGPU_MEM(T) resultsDevice = context.Malloc<T>(numRows);
		
	std::auto_ptr<SegReducePreprocessData> preprocessData;
	SegReduceCsrPreprocess<T>(count, csrDevice->get(), numRows, supportEmpty,
		&preprocessData, context);
	
	context.Start();
	for(int it = 0; it < numIterations; ++it) {
		if(TestTypeNormal == testType) 
			SegReduceCsr(dataDevice->get(), csrDevice->get(), count, numRows,
				supportEmpty, resultsDevice->get(), (T)0, mgpu::plus<T>(), 
				context);
		else if(TestTypeIndirect == testType) 
			IndirectReduceCsr(dataDevice->get(), csrDevice->get(),
				sourcesDevice->get(), count, numRows, supportEmpty,
				resultsDevice->get(), (T)0, mgpu::plus<T>(), context);
		else
			SegReduceApply(*preprocessData, dataDevice->get(), (T)0, 
				mgpu::plus<T>(), resultsDevice->get(), context);
	}
	double elapsed = context.Split();
	double throughput = (double)numIterations * count / elapsed;

	printf("%9.3lf M/s  %9.3lf GB/s\n", throughput / 1.0e6,
		sizeof(T) * throughput / 1.0e9);
	
	std::vector<T> resultsHost;
	resultsDevice->ToHost(resultsHost);

	std::vector<T> resultsRef(numRows);
	for(int row = 0; row < numRows; ++row) {
		int begin = csrHost[row]; 
		int end = (row + 1 < numRows) ? csrHost[row + 1] : count;
		int count = end - begin;
		
		begin = (TestTypeIndirect == testType) ? sourcesHost[row] : begin;
		end = begin + count;
	
		T x = 0;
		for(int i = begin; i < end; ++i)
			x = x + dataHost[i];
		
		resultsRef[row] = x;
	}
	
	for(int i = 0; i < numRows; ++i)
		if(resultsRef[i] != resultsHost[i]) {
			printf("REDUCTION ERROR ON SEGMENT %d\n", i);
			exit(0);
		}
}

const int Tests[][2] = { 
	{ 10000, 10000 },
	{ 50000, 10000 },
	{ 100000, 10000 },
	{ 200000, 5000 },
	{ 500000, 2000 },
	{ 1000000, 2000 },
	{ 2000000, 2000 },
	{ 5000000, 2000 },
	{ 10000000, 1000 },
	{ 20000000, 1000 }
};
const int NumTests = sizeof(Tests) / sizeof(*Tests); 

const int SegSizes[] = { 
	10,
	20,
	50,
	100,
	200,
	500,
	1000,
	2000,
	5000,
	10000,
	20000,
	50000,
	100000,
	200000,
	500000
};
const int NumSegSizes = sizeof(SegSizes) / sizeof(*SegSizes);

template<typename T>
void BenchmarkSegReduce1(TestType testType, bool supportEmpty, 
	CudaContext& context) {
	int avSegSize = 500;
	
	const char* typeString;
	if(TestTypeNormal == testType) typeString = "seg";
	else if(TestTypeIndirect == testType) typeString = "indirect";
	else typeString = "preprocess";

	printf("Benchmarking %s-reduce type %s. AvSegSize = %d.\n",
		typeString, TypeIdName<T>(), avSegSize);
	 
	for(int test = 0; test < NumTests; ++test) {
		int count = Tests[test][0];

		printf("%8s: ", FormatInteger(count).c_str());
		TestCsrReduce<T>(count, 2 * avSegSize, Tests[test][1], testType,
			supportEmpty, context);

		context.GetAllocator()->Clear();
	}
	printf("\n");
}

template<typename T>
void BenchmarkSegReduce2(TestType testType, bool supportEmpty, 
	CudaContext& context) {

	int count = 20000000;
	int numIterations = 500;
	
	const char* typeString;
	if(TestTypeNormal == testType) typeString = "seg";
	else if(TestTypeIndirect == testType) typeString = "indirect";
	else typeString = "preprocess";

	printf("Benchmarking %s-reduce type %s. Count = %d.\n",
		typeString, TypeIdName<T>(), count);
	
	for(int test = 0; test < NumSegSizes; ++test) {
		int avSegSize = SegSizes[test];
		
		printf("%8s: ", FormatInteger(avSegSize).c_str());
		TestCsrReduce<T>(count, 2 * avSegSize, numIterations, testType,
			supportEmpty, context);
		
		context.GetAllocator()->Clear();
	}
	printf("\n");
}

int main(int argc, char** argv) {
	ContextPtr context = CreateCudaDevice(argc, argv, true);

	bool supportEmpty = false;
	TestType testType = TestTypeNormal;

	BenchmarkSegReduce1<float>(testType, supportEmpty, *context);
	BenchmarkSegReduce1<double>(testType, supportEmpty, *context);

	BenchmarkSegReduce2<float>(testType, supportEmpty, *context);
	BenchmarkSegReduce2<double>(testType, supportEmpty,  *context);

	return 0;
}  