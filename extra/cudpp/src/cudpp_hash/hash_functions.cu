#include "hip/hip_runtime.h"
#include "hash_table.h"
#include "debugging.h"

#include <mt19937ar.h>

#include <cassert>

namespace CudaHT {
namespace CuckooHashing {

void GenerateFunctions(const unsigned  N,
                       const unsigned  num_keys,
                       const unsigned *d_keys,
                       const unsigned  table_size,
                             uint2    *constants) {
  bool regenerate = true;

  while (regenerate) {
    regenerate = false;

    // Generate a set of hash function constants for this build attempt.
    for (unsigned i = 0 ; i < N; ++i) {
      unsigned new_a = genrand_int32() % kPrimeDivisor;
      constants[i].x = (1 > new_a ? 1 : new_a);
      constants[i].y = genrand_int32() % kPrimeDivisor;
    }

#ifdef FORCEFULLY_GENERATE_NO_CYCLES
    // Ensure that every key gets N different slots.
    regenerate = CheckAssignedSameSlot(N, num_keys, d_keys, table_size, constants);
#endif
  }


#ifdef TAKE_HASH_FUNCTION_STATISTICS
  // Examine how well distributed the items are.
  TakeHashFunctionStatistics(num_keys, d_keys, table_size, constants, N);
#endif
}

}; // namespace CuckooHashing
}; // namespace CudaHT
