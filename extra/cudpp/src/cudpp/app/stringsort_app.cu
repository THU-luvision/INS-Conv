// -------------------------------------------------------------
// CUDPP -- CUDA Data Parallel Primitives library
// -------------------------------------------------------------
// $Revision$
// $Date$
// -------------------------------------------------------------
// This source code is distributed under the terms of license.txt
// in the root directory of this source distribution.
// -------------------------------------------------------------

/**
* @file
* stringsort_app.cu
*
* @brief CUDPP application-level merge sorting routines
*/

/** @addtogroup cudpp_app
* @{
*/

/** @name StringSort Functions
* @{
*/

#include "cuda_util.h"
#include "cudpp.h"
#include "cudpp_util.h"
#include "cudpp_stringsort.h"
#include "kernel/stringsort_kernel.cuh"
#include "limits.h"


#define BLOCKSORT_SIZE 1024
#define DEPTH 8

void dotAdd(unsigned int* d_address,
           unsigned int* numSpaces,
           unsigned int* packedAddress,
           size_t numElements,
           size_t stringArrayLength)
{
        int numThreads = 128;
        int numBlocks = (numElements+numThreads-1)/numThreads;
        dotAddInclusive<<<numBlocks, numThreads>>>(numSpaces, d_address, packedAddress, numElements, stringArrayLength);
}
void calculateAlignedOffsets(unsigned int* d_address,
                                                         unsigned int* numSpaces,
                                                         unsigned char* d_stringVals,
                                                         unsigned char termC,
                                                         size_t numElements,
                                                         size_t stringArrayLength)
{
        int numThreads = 128;
        int numBlocks = (numElements+numThreads-1)/numThreads;

        alignedOffsets<<<numBlocks, numThreads>>>(numSpaces, d_address, d_stringVals, termC, numElements, stringArrayLength);

}
void packStrings(unsigned int* packedStrings,
                                                 unsigned char* d_stringVals,
                                                 unsigned int* d_keys,
                                                 unsigned int* packedAddress,
                                                 unsigned int* address,
                                                 size_t numElements,
                                                 size_t stringArrayLength,
                                                 unsigned char termC)
{
        unsigned int numThreads = 128;
        unsigned int numBlocks = (numElements + numThreads - 1)/numThreads;

        //Each thread handles one string (irregular parrallelism) other option is to do per character (set of chars)
        //but that requires a binary search per character. Efficiency depends on the dataset
        alignString<<<numBlocks, numThreads>>>(packedStrings, d_stringVals, packedAddress, address, numElements, stringArrayLength, termC);
        createKeys<<<numBlocks, numThreads>>>(d_keys, packedStrings, packedAddress, numElements);

}


void unpackStrings(unsigned int* packedAddress,
                                   unsigned int* packedAddressRef,
                                   unsigned int* address,
                                   unsigned int* addressRef,
                                   size_t numElements)
{
        unsigned int numThreads = 128;
        unsigned int numBlocks = (numElements + numThreads - 1)/numThreads;

        unpackAddresses<<<numBlocks, numThreads>>>(packedAddress, packedAddressRef, address, addressRef, numElements);
}

/** @brief Performs merge sor utilzing three stages.
* (1) Blocksort, (2) simple merge and (3) multi merge on a
* set of strings
*
* @param[in,out] pkeys Keys (first four characters of string) to be sorted.
* @param[in,out] pvals Addresses of string locations for tie-breaks
* @param[out] stringVals global string value array (four characters stuffed into a uint)
* @param[in] numElements Number of elements in the sort.
* @param[in] stringArrayLength The size of our string array in uints (4 chars per uint)
* @param[in] plan Configuration information for mergesort.
* @param[in] termC Termination character for our strings
**/
void runStringSort(unsigned int *pkeys,
                                   unsigned int *pvals,
                                   unsigned int *stringVals,
                                   size_t numElements,
                                   size_t stringArrayLength,
                                   unsigned char termC,
                                   const CUDPPStringSortPlan *plan)
{
        int numPartitions = (numElements+BLOCKSORT_SIZE-1)/BLOCKSORT_SIZE;
        int numBlocks = numPartitions/2;
        int partitionSize = BLOCKSORT_SIZE;




        unsigned int swapPoint = plan->m_swapPoint;
        unsigned int subPartitions = plan->m_subPartitions;



        int numThreads = 128;

        blockWiseStringSort<unsigned int, DEPTH> <<<numPartitions, BLOCKSORT_SIZE/DEPTH, 2*(BLOCKSORT_SIZE)*sizeof(unsigned int)>>>
                                     (pkeys, pvals, stringVals, BLOCKSORT_SIZE, numElements, stringArrayLength, termC);


        int mult = 1; int count = 0;

        //we run p stages of simpleMerge until numBlocks <= some Critical level
        while(numPartitions > swapPoint || (partitionSize*mult < 16384 && numPartitions > 1)/* && numPartitions > 1*/)
        {
                //printf("Running simple merge for %d partitions of size %d\n", numPartitions, partitionSize*mult);
                numBlocks = (numPartitions&0xFFFE);
                if(count%2 == 0)
                {
                        simpleStringMerge<unsigned int, 2>
                                <<<numBlocks, CTASIZE_simple, sizeof(unsigned int)*(2*INTERSECT_B_BLOCK_SIZE_simple+4)>>>(pkeys, plan->m_tempKeys,
                                pvals, plan->m_tempAddress, stringVals, partitionSize*mult, numElements, count, stringArrayLength, termC);

                        if(numPartitions%2 == 1)
                        {

                                int offset = (partitionSize*mult*(numPartitions-1));
                                int numElementsToCopy = numElements-offset;
                                simpleCopy<unsigned int>
                                        <<<(numElementsToCopy+numThreads-1)/numThreads, numThreads>>>(pkeys, pvals, plan->m_tempKeys, plan->m_tempAddress, offset, numElementsToCopy);
                        }
                }
                else
                {
                        simpleStringMerge<unsigned int, 2>
                                <<<numBlocks, CTASIZE_simple, sizeof(unsigned int)*(2*INTERSECT_B_BLOCK_SIZE_simple+4)>>>(plan->m_tempKeys, pkeys,
                                plan->m_tempAddress, pvals, stringVals, partitionSize*mult, numElements, count, stringArrayLength, termC);

                        if(numPartitions%2 == 1)
                        {
                                int offset = (partitionSize*mult*(numPartitions-1));
                                int numElementsToCopy = numElements-offset;
                                simpleCopy<unsigned int>
                                        <<<(numElementsToCopy+numThreads-1)/numThreads, numThreads>>>(plan->m_tempKeys, plan->m_tempAddress, pkeys, pvals, offset, numElementsToCopy);
                        }
                }

                mult*=2;
                count++;
                numPartitions = (numPartitions+1)/2;
        }




        //End of simpleMerge, now blocks cooperate to merge partitions
        while (numPartitions > 1)
        {
                numBlocks = (numPartitions&0xFFFE);
                int secondBlocks = ((numBlocks)*subPartitions+numThreads-1)/numThreads;
                if(count%2 == 1)
                {
                        findMultiPartitions<unsigned int>
                                <<<secondBlocks, numThreads>>>(plan->m_tempKeys, plan->m_tempAddress, stringVals, subPartitions, numBlocks, partitionSize*mult, plan->m_partitionStartA, plan->m_partitionSizeA,
                                plan->m_partitionStartB, plan->m_partitionSizeB, numElements, stringArrayLength, termC);


                        //int lastSubPart = getLastSubPart(numBlocks, subPartitions, partitionSize, mult, numElements);
                        stringMergeMulti<unsigned int, DEPTH_multi>
                                <<<numBlocks*subPartitions, CTASIZE_multi, (2*INTERSECT_B_BLOCK_SIZE_multi+4)*sizeof(unsigned int)>>>(plan->m_tempKeys, pkeys, plan->m_tempAddress,
                                pvals, stringVals, subPartitions, numBlocks, plan->m_partitionStartA, plan->m_partitionSizeA, plan->m_partitionStartB, plan->m_partitionSizeB, mult*partitionSize,
                                count, numElements, stringArrayLength, termC);
                        if(numPartitions%2 == 1)
                        {
                                int offset = (partitionSize*mult*(numPartitions-1));
                                int numElementsToCopy = numElements-offset;
                                simpleCopy<unsigned int>
                                        <<<(numElementsToCopy+numThreads-1)/numThreads, numThreads>>>(plan->m_tempKeys, plan->m_tempAddress, pkeys, pvals, offset, numElementsToCopy);
                        }

                }
                else
                {

                        findMultiPartitions<unsigned int>
                                <<<secondBlocks, numThreads>>>(pkeys, pvals, stringVals, subPartitions, numBlocks, partitionSize*mult, plan->m_partitionStartA, plan->m_partitionSizeA,
                                plan->m_partitionStartB, plan->m_partitionSizeB, numElements, stringArrayLength, termC);
                        //int lastSubPart = getLastSubPart(numBlocks, subPartitions, partitionSize, mult, numElements);
                        stringMergeMulti<unsigned int, DEPTH_multi>
                                <<<numBlocks*subPartitions, CTASIZE_multi, (2*INTERSECT_B_BLOCK_SIZE_multi+4)*sizeof(unsigned int)>>>(pkeys, plan->m_tempKeys, pvals,
                                plan->m_tempAddress, stringVals, subPartitions, numBlocks, plan->m_partitionStartA, plan->m_partitionSizeA, plan->m_partitionStartB, plan->m_partitionSizeB, mult*partitionSize,
                                count, numElements, stringArrayLength, termC);

                        if(numPartitions%2 == 1)
                        {
                                int offset = (partitionSize*mult*(numPartitions-1));
                                int numElementsToCopy = numElements-offset;
                                simpleCopy<unsigned int>
                                        <<<(numElementsToCopy+numThreads-1)/numThreads, numThreads>>>(pkeys, pvals, plan->m_tempKeys, plan->m_tempAddress, offset, numElementsToCopy);
                        }

                }


                count++;
                mult*=2;
                subPartitions*=2;
                numPartitions = (numPartitions+1)/2;
        }

        if(count%2==1)
        {
                CUDA_SAFE_CALL(hipMemcpy(pkeys, plan->m_tempKeys, numElements*sizeof(unsigned int), hipMemcpyDeviceToDevice));
                CUDA_SAFE_CALL(hipMemcpy(pvals, plan->m_tempAddress, numElements*sizeof(unsigned int), hipMemcpyDeviceToDevice));
        }

}

#ifdef __cplusplus
extern "C"
{
#endif


        /**
        * @brief From the programmer-specified sort configuration,
        *        creates internal memory for performing the sort.
        *
        * @param[in] plan Pointer to CUDPPStringSortPlan object
        **/
        void allocStringSortStorage(CUDPPStringSortPlan *plan)
        {


                CUDA_SAFE_CALL(hipMalloc((void**)&plan->m_keys, sizeof(unsigned int)*plan->m_numElements));
                CUDA_SAFE_CALL(hipMalloc((void**)&plan->m_tempKeys,    sizeof(unsigned int)*plan->m_numElements));
                CUDA_SAFE_CALL(hipMalloc((void**)&plan->m_tempAddress,    sizeof(unsigned int)*plan->m_numElements));
                CUDA_SAFE_CALL(hipMalloc((void**)&plan->m_packedAddress, sizeof(unsigned int)*(plan->m_numElements+1)));
                CUDA_SAFE_CALL(hipMalloc((void**)&plan->m_packedAddressRef, sizeof(unsigned int)*(plan->m_numElements)));
                CUDA_SAFE_CALL(hipMalloc((void**)&plan->m_addressRef, sizeof(unsigned int)*(plan->m_numElements)));

                CUDA_SAFE_CALL(hipMalloc((void**)&plan->m_spaceScan, sizeof(unsigned int)*(plan->m_numElements+1)));
                CUDA_SAFE_CALL(hipMalloc((void**)&plan->m_numSpaces, sizeof(unsigned int)*(plan->m_numElements+1)));

                CUDA_SAFE_CALL(hipMalloc((void**)&plan->m_partitionSizeA, sizeof(unsigned int)*(plan->m_swapPoint*plan->m_subPartitions*4)));
                CUDA_SAFE_CALL(hipMalloc((void**)&plan->m_partitionSizeB, sizeof(unsigned int)*(plan->m_swapPoint*plan->m_subPartitions*4)));
                CUDA_SAFE_CALL(hipMalloc((void**)&plan->m_partitionStartA, sizeof(unsigned int)*(plan->m_swapPoint*plan->m_subPartitions*4)));
                CUDA_SAFE_CALL(hipMalloc((void**)&plan->m_partitionStartB, sizeof(unsigned int)*(plan->m_swapPoint*plan->m_subPartitions*4)));
        }

        /** @brief Deallocates intermediate memory from allocStringSortStorage.
        *
        *
        * @param[in] plan Pointer to CUDPStringSortPlan object
        **/

        void freeStringSortStorage(CUDPPStringSortPlan* plan)
        {
                hipFree(plan->m_keys);
                hipFree(plan->m_packedAddress);
                hipFree(plan->m_packedAddressRef);
                hipFree(plan->m_tempKeys);
                hipFree(plan->m_tempAddress);
                hipFree(plan->m_addressRef);

                hipFree(plan->m_numSpaces);
                hipFree(plan->m_spaceScan);

                hipFree(plan->m_partitionSizeA);
                hipFree(plan->m_partitionSizeB);
                hipFree(plan->m_partitionStartA);
                hipFree(plan->m_partitionStartB);
        }

        /** @brief Dispatch function to perform a sort on an array with
        * a specified configuration.
        *
        * This is the dispatch routine which calls stringSort...() with
        * appropriate template parameters and arguments as specified by
        * the plan.
        * @param[in,out] keys Keys (first four chars of string) to be sorted.
        * @param[in,out] values Address of string values in array of null terminated strings
        * @param[in] stringVals Global string array
        * @param[in] numElements Number of elements in the sort.
        * @param[in] stringArrayLength The size of our string array in uints (4 chars per uint)
        * @param[in] termC Termination character for our strings
        * @param[in] plan Configuration information for mergeSort.
        **/

        void cudppStringSortDispatch(unsigned int  *keys,
                                         unsigned int  *values,
                                         unsigned int  *stringVals,
                                         size_t numElements,
                                                                 size_t stringArrayLength,
                                                                 unsigned char termC,
                                         const CUDPPStringSortPlan *plan)
        {
                runStringSort(keys, values, stringVals, numElements, stringArrayLength, termC, plan);
        }

#ifdef __cplusplus
}
#endif






/** @} */ // end stringsort functions
/** @} */ // end cudpp_app
