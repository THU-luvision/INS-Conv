// -------------------------------------------------------------
// CUDPP -- CUDA Data Parallel Primitives library
// -------------------------------------------------------------
// $Revision: 4400 $
// $Date: 2008-08-04 10:58:14 -0700 (Mon, 04 Aug 2008) $
// ------------------------------------------------------------- 
// This source code is distributed under the terms of license.txt 
// in the root directory of this source distribution.
// ------------------------------------------------------------- 

/**
 * @file
 * tridiagonal_app.cu
 *
 * @brief CUDPP application-level tridiagonal solver routines
 */

/** \addtogroup cudpp_app
  * @{
  */
/** @name Tridiagonal functions
 * @{
 */

#include "cudpp.h"
#include "cudpp_util.h"
#include "cudpp_plan.h"
#include "cudpp_manager.h"
#include "cuda_util.h"

#include <cstdlib>
#include <cstdio>
#include <assert.h>

#include "kernel/tridiagonal_kernel.cuh"

template <typename T>
inline unsigned int crpcrSharedSize(unsigned int systemSizeOriginal)
{
    const unsigned int systemSize = ceilPow2(systemSizeOriginal);
    const unsigned int restSystemSize = systemSize/2;
    return (systemSize + 1 + restSystemSize) * 5 * sizeof(T);
}

/**
 * @brief Hybrid CR-PCR solver (CRPCR)
 *
 * This is a wrapper function for the GPU CR-PCR kernel.
 *
 * @param[out] d_x Solution vector
 * @param[in] d_a Lower diagonal
 * @param[in] d_b Main diagonal
 * @param[in] d_c Upper diagonal
 * @param[in] d_d Right hand side
 * @param[in] systemSizeOriginal The size of the linear system
 * @param[in] numSystems The number of systems to be solved
 */
template <typename T>
void crpcr(T *d_a, 
           T *d_b, 
           T *d_c, 
           T *d_d, 
           T *d_x, 
           unsigned int systemSizeOriginal, 
           unsigned int numSystems)
{
    const unsigned int systemSize = ceilPow2(systemSizeOriginal);
    const unsigned int num_threads_block = systemSize/2;
    const unsigned int restSystemSize = systemSize/2;
    const unsigned int iterations = logBase2Pow2(restSystemSize/2);
  
    // setup execution parameters
    dim3  grid(numSystems, 1, 1);
    dim3  threads(num_threads_block, 1, 1);
    const unsigned int smemSize = crpcrSharedSize<T>(systemSizeOriginal);

    crpcrKernel<<< grid, threads, smemSize>>>(d_a, 
                                              d_b, 
                                              d_c, 
                                              d_d, 
                                              d_x, 
                                              systemSizeOriginal,
                                              iterations);

    CUDA_CHECK_ERROR("crpcr");
}


/**
 * @brief Dispatches the tridiagonal function based on the plan
 *
 * This is the dispatch call for the tridiagonal solver in either float 
 * or double datatype. 
 *
 * @param[out] d_x Solution vector
 * @param[in] d_a Lower diagonal
 * @param[in] d_b Main diagonal
 * @param[in] d_c Upper diagonal
 * @param[in] d_d Right hand side
 * @param[in] systemSize The size of the linear system
 * @param[in] numSystems The number of systems to be solved
 * @param[in] plan pointer to CUDPPTridiagonalPlan
 * @returns CUDPPResult indicating success or error condition
 */
CUDPPResult cudppTridiagonalDispatch(void *d_a, 
                                     void *d_b, 
                                     void *d_c, 
                                     void *d_d, 
                                     void *d_x, 
                                     int systemSize, 
                                     int numSystems, 
                                     const CUDPPTridiagonalPlan * plan)
{
    hipDeviceProp_t prop;
    plan->m_planManager->getDeviceProps(prop);

    if (ceilPow2(systemSize) > (unsigned)prop.maxThreadsPerBlock)
        return CUDPP_ERROR_ILLEGAL_CONFIGURATION;

    //figure out which algorithm to run
    if (plan->m_config.datatype == CUDPP_FLOAT)
    {
        // check necessary memory
        if (crpcrSharedSize<float>(systemSize) > prop.sharedMemPerBlock)
            return CUDPP_ERROR_INSUFFICIENT_RESOURCES;

        crpcr<float>((float *)d_a, 
                     (float *)d_b, 
                     (float *)d_c, 
                     (float *)d_d, 
                     (float *)d_x, 
                     systemSize, 
                     numSystems);
        return CUDPP_SUCCESS;
    }
    else if (plan->m_config.datatype == CUDPP_DOUBLE)
    {
        // check necessary memory
        if (crpcrSharedSize<double>(systemSize) > prop.sharedMemPerBlock)
            return CUDPP_ERROR_INSUFFICIENT_RESOURCES;

        crpcr<double>((double *)d_a, 
                      (double *)d_b, 
                      (double *)d_c, 
                      (double *)d_d, 
                      (double *)d_x, 
                      systemSize, 
                      numSystems);
        return CUDPP_SUCCESS;
    }
    else
        return CUDPP_ERROR_ILLEGAL_CONFIGURATION;
    
}

/** @} */ // end Tridiagonal functions
/** @} */ // end cudpp_app
